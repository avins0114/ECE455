#include "hip/hip_runtime.h"
#include "helpers.h"

// ==================== Coalesced Kernel ====================

template <typename T>
__global__ void mm_coalesced_kernel(T const* mat_1, T const* mat_2, T* mat_3,
                                    size_t m, size_t n, size_t p)
{
    // Swapped x/y mapping for better memory coalescing
    // Columns -> x dimension, rows -> y dimension
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};

    // Do not process outside the matrix.
    if ((i >= m) || (j >= p))
    {
        return;
    }

    T acc_sum{0};
    for (size_t k{0}; k < n; ++k)
    {
        acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
    }
    mat_3[i * p + j] = acc_sum;
}

// ==================== Kernel Launcher ====================

template <typename T>
void mm_cuda(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n,
             size_t p)
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    mm_coalesced_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, m, n,
                                                                p);
}

// ==================== Main ====================

int main()
{
    const size_t num_tests{2};

    assert(random_multiple_test_mm_cuda<int32_t>(num_tests));
    assert(random_multiple_test_mm_cuda<float>(num_tests));
    assert(random_multiple_test_mm_cuda<double>(num_tests));

    std::cout << "All tests passed!\n";

    const size_t num_measurement_tests{2};
    const size_t num_measurement_warmups{1};
    size_t m{MAT_DIM}, n{MAT_DIM}, p{MAT_DIM};

    float mm_cuda_int32_latency = measure_latency_mm_cuda<int32_t>(
        m, n, p, num_measurement_tests, num_measurement_warmups);
    float mm_cuda_float_latency = measure_latency_mm_cuda<float>(
        m, n, p, num_measurement_tests, num_measurement_warmups);
    float mm_cuda_double_latency = measure_latency_mm_cuda<double>(
        m, n, p, num_measurement_tests, num_measurement_warmups);

    std::cout << "Matrix Multiplication Runtime\n";
    std::cout << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << "\n";
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << mm_cuda_int32_latency << " ms\n";
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << mm_cuda_float_latency << " ms\n";
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << mm_cuda_double_latency << " ms\n";

    return 0;
}
